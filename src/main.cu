#include "hip/hip_runtime.h"
// This will be the main C file to play with
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <strings.h>
#include <errno.h>
#include "helpers.cu.h"
#include "cuda-kernels.cu.h"
#include "cpu-kernels.cu.h"

int validate(){
    return 0;
}

int readDataset(char* pathname, dataset* ds){
// Read a dataset in the newline seperated format:
// trend - i32
// k - i32
// n - i32
// freq - f32
// hfrac - f32
// lam - f32
// N - i32
// m - i32
// mappingindices - i32[N]
// images - f32[m][N]
 
    FILE* file = fopen(pathname, "r");
    if (file == NULL){
        printf("Error reading dataset %s:\n%s\n", pathname, strerror(errno));
        return 1;
    }
    // Init dataset struct
    fscanf(file, "%d\n%d\n%d\n%f\n%f\n%f\n%d\n%d\n",
            &ds->trend, &ds->k, &ds->n,
            &ds->freq, &ds->hfrac, &ds->lam,
            &ds->N, &ds->m
        );

    // Read mappingIndices
    ds->mappingIndices = (int*)malloc(ds->N*sizeof(int));
    int e = readIntArray(ds->N, ds->mappingIndices, file);
    switch(e){
        case 0:
            printf("Successfully read Dataset mappingindices\n");
            break;
        case -1:
            printf("FP not pointing to start of array\n");
            return -1;
        case -2:
            printf("Error in scanning array\n");
            return -1;
        case -3:
            printf("Error in scanning end of array\n");
            return -1;
        default:
            printf("Unexpected error in array scan\n");
            return -1;
    }

    // Read the image array
    ds->images = (float*) malloc(ds->m*ds->N*sizeof(float));
    float* curImage = ds->images;
    char first = (char) fgetc(file);
    if (first != '['){
        printf("Error in reading matrix start\n");
        return -1;
    }
    for(int i = 0; i < ds->m; i++){
        e = readFloatArray(ds->N, curImage, file);
        switch(e){
            case 0:
                break;
            case -1:
                printf("FP not pointing to start of array\n");
                return -1;
            case -2:
                printf("Error in scanning array\n");
                return -1;
            case -3:
                printf("Error in scanning end of array\n");
                return -1;
            default:
                printf("Unexpected error in array scan\n");
                return -1;
        }
        curImage += ds->N;
        if (i != ds->m - 1 && ((char)fgetc(file) != ',' || (char)fgetc(file) != ' ')){
            printf("Format Error\n");
            return -1;
        }
    }
    printf("Successfully read Image Array\n");

    fclose(file);
    return 0;
}
int validate(dataset* ds){
    // KERNEL 1
    // Make interpolation matrix
    printf("Creating X matrix\n");
    int k2p2 = 2*ds->k + 2;
    int k2p2_ = (ds->trend > 0) ? k2p2 : k2p2-1;
    float* X_host = (float*) malloc(k2p2_ * ds->N * sizeof(float));
    seq_mkX(k2p2_, ds->N, ds->freq, ds->mappingIndices, X_host);

    printf("Transposing matrices and extracting Historical data\n");
    float* Xh_host = (float*) malloc(k2p2_ * ds->n * sizeof(float));
    float* Xth_host= (float*) malloc(k2p2_ * ds->n * sizeof(float));
    float* Yh_host = (float*) malloc(ds->m * ds->n * sizeof(float));
    // Do the list slicing sequentially
    for(int k = 0; k < k2p2_; k++){
        for (int i = 0; i < ds->n; i++){
            // Copy Xh[:,:n]
            Xh_host[k*ds->n + i] = X_host[k*ds->N + i];
        }
    }
    for(int j = 0; j < ds->m; j++){
        for (int i = 0; i < ds->n; i++){
            // Copy Yh[:,:n]
            Yh_host[j*ds->n + i] = ds->images[j*ds->N + i];
        }
    }
    // Transpose X
    seq_transpose(Xh_host, Xth_host, k2p2_, ds->n);
    printMatrix(Xth_host,  ds-> n, k2p2_);
    printf("[!]K1 done\n");

    // KERNEL 2
    printf("Creating Xsqr\n");
    float* Xsqr_host = (float*) malloc(k2p2_ * k2p2_ * sizeof(float));
    seq_mmMulFilt(Xh_host, Xth_host, Yh_host, Xsqr_host, ds->n, k2p2_, k2p2);
    printf("[!]K2 Done\n");
    // KERNEL 3
    printf("Inverting Xsqr\n");
    float* Xinv_host = (float*) malloc(k2p2_ * k2p2_ * sizeof(float));
    seq_matInv(Xsqr_host, Xinv_host, k2p2_);
    printf("[!]K3 Done\n");

    // Kernel 4
    printf("running Vector Multiplication and calculating betas\n");
    printf("Filtered first\n");
    float* beta0_host = (float*) malloc(k2p2_ * ds->m * sizeof(float));
    for(int row = 0; row < ds->m; row++){
        seq_mvMulFilt(Xh_host, (Yh_host + (row*ds->n)), beta0_host+(row*k2p2_), k2p2, ds->n);
    }

    printf("Unfiltered beta and y_preds\n");
    float* beta_host = (float*) malloc(k2p2_ * ds->m * sizeof(float));
    for(int row = 0; row < ds->m; row++){
        seq_mvMul(Xinv_host, (beta0_host + (row*k2p2_)), beta_host+(row*k2p2_), k2p2_, k2p2_);
    }
    float* ypreds_host = (float*) malloc(ds->N * ds->m * sizeof(float));
    // Transpose X
    float* Xt_host = (float*) malloc(ds->N * k2p2_ * sizeof(float));
    seq_transpose(X_host, Xt_host, k2p2_, ds->N);
    
    for(int row = 0; row < ds->m; row++){
        seq_mvMul(Xt_host, beta_host + (row*k2p2_), ypreds_host + (row*ds->N), ds->N, k2p2_);
    }
    printf("[!]K4 Done\n");
    // Kernel 5
    printf("Calculating Y_errors\n");
    float* r_host = (float*) malloc(ds->m * ds->N * sizeof(float));
    seq_YErrorCalculation(ds->images, ypreds_host, r_host, ds->m, ds->N);
    printf("[!]K5 Done\n");
    
    // Kernel 6
    printf("Calculating Sigmas\n");
    float* sigmas_host = (float*) malloc(ds->m * sizeof(float));
    float* ns_host     = (float*) malloc(ds->m * sizeof(float));
    int* hs_host     = (int*) malloc(ds->m * sizeof(float));
    seq_NSSigma(r_host, Yh_host, sigmas_host, hs_host, ns_host, ds->N, ds->m, k2p2_, ds->hfrac);
    printf("Sigmas calculated\n");
    printf("[!]K6 Done\n");

    // Kernel 7
    printf("Calculating hmax: ");
    int hmax = -100000;
    for(int i = 0; i < ds->m; i++){
        if (hs_host[i] > hmax)
            hmax = hs_host[i];
    } 
    printf("%d\n", hmax);
    float* MOfst_host = (float*) malloc(ds->m * sizeof(float*));
    float* BOUND_host = (float*) malloc((ds->N - ds->n)*sizeof(float));
    seq_msFst(hmax, r_host, hs_host, ns_host, MOfst_host, BOUND_host, ds->N, ds->n);
    printf("Calculated MO_fsts and Bounds\n");
    printf("[!]K7 Done\n");

    float* breaks_host = (float*) malloc(ds->m * sizeof(float*));
    float* means_host  = (float*) malloc(ds->m * sizeof(float*));
    
    // Moving sums
    for(int pixel = 0; pixel < ds->m; pixel++){
        float* _pixel = ds->images + (pixel*ds->m);
        float* MO = (float*) malloc((ds->N - ds->n) * sizeof(float));
        // Calculate SUM^t_
        float sigSqrN = sigmas_host[pixel] * sqrtf(ns_host[pixel]);
        for(int i = 0; i < ds->N - ds->n; i++){
            if (i == 0){ // Put in mo_fst
                MO[i] = MOfst_host[pixel];
            }
            // I'm not sure this needs to be there, since we allow nan in MO
            //if(isnan(_pixel[i]))
                //MO[i] = MO[i-1]; // If Nan, assume no forest cut down in the meantime
                //continue;
            int h = hs_host[pixel];
            // r[n + t] - r[n + t - h]
            float tmp = (r_host[pixel*ds->m + ds->n + i] - r_host[pixel*ds->m + (ds->n + i - h)]);
            MO[i] = tmp / sigSqrN;

        }
        // Now MO is calculated
        int firstBreak = -1;
        for (int j = 0; j < ds->N - ds->n; j++){
            float b = BOUND_host[j];
            float mo= MO[j];
            if (!isnan(mo)){
                if (fabsf(mo) > b){
                    firstBreak = j;
                    break;
                }
            }
        }
        breaks_host[pixel] = firstBreak;
    }

    // Free everything!!!
    // TODO:
    free(X_host);
    free(Xh_host);
    return 0;
    
}


int main(int argc, char* argv[]){
    float* a = (float*)malloc(8*sizeof(float));
    float* b = (float*)malloc(8*sizeof(float));
    for(int i = 0; i < 4; i++){
        for(int j = 0; j < 2; j++){
            a[i*2+j] = (float)(i+j);
        }
    }
    printMatrix(a, 4, 2);
    for(int i = 0; i < 2; i++){
        for(int j = 0; j < 4; j++){
            b[i*4+j] = (float)(i+j);
        }
    }
    printMatrix(b, 2, 4);
    float* c = (float*)malloc(4*4*sizeof(float));
    float* idk=(float*)malloc(1024*sizeof(float));
    seq_mmMulFilt(a, b, idk, c, 4, 2, 4);
    printMatrix(c, 4, 4);

    return 0;
    dataset* ds = (dataset*) malloc(sizeof(dataset));
    char* dsPath = "data/small_peru.clean";
    readDataset(dsPath, ds);
    printf("Ready to work on dataset of %d images, with %d pixels each\n", ds->N, 
            ds->m);
    validate(ds);
    return 0;
}
